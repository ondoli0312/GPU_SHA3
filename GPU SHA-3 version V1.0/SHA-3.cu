#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "keccak.h"
#include "hip/hip_runtime.h"
#include ""


static void HandleError(hipError_t err,
	const char* file,
	int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

/*
__device__ unsigned char offset[25] = {
	0, 36, 3, 41, 18,
	1, 44, 10, 45, 2,
	62, 6, 43, 15, 61,
	28, 55, 25, 21, 56,
	27, 20, 39, 8, 14
};
*/

//__global__ void test(ulong* state, ulong* BC, unsigned char* offset, ulong* out)
__global__ void test( ulong* out)
{
	ulong state[25] = {0x0,};
	ulong BC[25] = { 0x0, };
	unsigned char offset[25] = { 0x0, };


	__shared__ ulong temp[25];


	int tid = threadIdx.x;
	ulong temp1 = BC[(tid + 4) % 5] ^ (ROTL64((BC[tid + 1] % 5), 1));

	temp[tid] = ROTL64(state[tid] ^ temp1, offset[tid * 5]);
	temp[tid + 5] = ROTL64(state[tid + 5] ^ temp1, offset[tid * 5 + 1]);
	temp[tid + 10] = ROTL64(state[tid + 10] ^ temp1, offset[tid * 5 + 2]);
	temp[tid + 15] = ROTL64(state[tid + 15] ^ temp1, offset[tid * 5 + 3]);
	temp[tid + 20] = ROTL64(state[tid + 20] ^ temp1, offset[tid * 5 + 4]);
	__syncthreads();
	for (int i = 0; i < 25; i++)
		out[i] = temp[i];
}


int main()
{

	ulong* output = (ulong*)malloc(sizeof(ulong) * 25);
	ulong* dev_output;
	ulong A[25];
	ulong BC[5];
	for (int i = 0; i < 25; i++)
		A[i] = i;
	
	for (int i = 0; i < 5; i++)
		BC[i] = INT_MAX + i;

	unsigned char offset[25] = {
	0, 36, 3, 41, 18,
	1, 44, 10, 45, 2,
	62, 6, 43, 15, 61,
	28, 55, 25, 21, 56,
	27, 20, 39, 8, 14
	};

	HANDLE_ERROR(hipMalloc((void**)&dev_output, sizeof(ulong)*25));
	//test << <1, 5 >> > (A, BC, offset,dev_output);
	test << <1, 5 >> > (dev_output);
	HANDLE_ERROR(hipMemcpy(output, dev_output, sizeof(ulong)*25, hipMemcpyDeviceToHost));


	//HANDLE_ERROR(hipMemcpy(output, dev_output, 25, hipMemcpyDeviceToHost));
	
	for (int i = 0; i < 25; i++)
		printf("%llx ", output[i]);
	
	/*
	
	int c[10];
	int b[10];
	int a[10];
	int* dev_c = NULL;
	int* dev_b = NULL;
	int* dev_a = NULL;
	
	HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int) * 10));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, sizeof(int) * 10));
	HANDLE_ERROR(hipMalloc((void**)&dev_a, sizeof(int) * 10));

	for (int i = 0; i < 10; i++) {
		b[i] = i + 1;
		a[i] = i + 2;
	}

	HANDLE_ERROR(hipMemcpy(dev_a, a, sizeof(int) * 10, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, sizeof(int) * 10, hipMemcpyHostToDevice));
	add<<<10,10>>>(dev_a, dev_b, dev_c);

	HANDLE_ERROR(hipMemcpy(c, dev_c, sizeof(int) * 10, hipMemcpyDeviceToHost));
	for (int i = 0; i < 10; i++)
		printf("%d ", c[i]);
	*/
}

#if 0 
ulong RC[Round] = {
   0x0000000100000000, 0x0000808200000000, 0x0000808a80000000, 0x8000800080000000,
   0x0000808b00000000, 0x8000000100000000, 0x8000808180000000, 0x0000800980000000,
   0x0000008a00000000, 0x0000008800000000, 0x8000800900000000, 0x8000000a00000000,
   0x8000808b00000000, 0x0000008b80000000, 0x0000808980000000, 0x0000800380000000,	
   0x0000800280000000, 0x0000008080000000, 0x0000800a00000000, 0x8000000a80000000,
   0x8000808180000000, 0x0000808080000000, 0x8000000100000000, 0x8000800880000000
};

ulong State[25] = {0, };
ulong BC[5] = { 0, };
ulong buffer[25] = { 0, };

const byte offset[Round + 1] = {
	0, 36, 3, 41, 18,
	1, 44, 10, 45, 2,
	62, 6, 43, 15, 61,
	28, 55, 25, 21, 56,
	27, 20, 39, 8, 14
};

typedef struct SHA {
	byte buffer[136];
	ulong msglen;
}SHA3_INFO;

struct Theta_PARAMS {
	ulong state[5];
	int flag;
	int theta;
};


void Pho_Theta_function_No_use_Thread(ulong theta, ulong flag, ulong state0, ulong state1, ulong state2, ulong state3, ulong state4)
{
	ulong temp1 = RC[(theta + 4) % 5] ^ (ROTL64(BC[(theta + 1) % 5], 1));
	buffer[theta] = ROTL64(state0 ^ temp1, offset[flag]);
	buffer[theta + 5] = ROTL64(state1 ^ temp1, offset[flag + 1]);
	buffer[theta + 10] = ROTL64(state2 ^ temp1, offset[flag + 2]);
	buffer[theta + 15] = ROTL64(state3 ^ temp1, offset[flag + 3]);
	buffer[theta + 20] = ROTL64(state4 ^ temp1, offset[flag + 4]);

}


void Chi_lota_function_No_use_Thread(ulong flag, ulong state0, ulong state1, ulong state2, ulong state3, ulong state4) {

	State[flag] = ((state0) ^ ((~(state1)) & state2));
	State[flag + 1] = ((state1) ^ ((~(state2)) & state3));
	State[flag + 2] = ((state2) ^ ((~(state3)) & state4));
	State[flag + 3] = ((state3) ^ ((~(state4)) & state0));
	State[flag + 4] = ((state4) ^ ((~(state0)) & state1));
}

void keecak_Function_NO_use_Thread(SHA3_INFO* info)
{
	int j = 0;
	for (int i = 0; i < 17; i++) {

		State[i] = State[i] ^ ENDIAN_CHANGE(*(ulong*)(info->buffer + (i << 3)));
	}

	for (j = 0; j < Round; j++)
	{

		BC[0] = State[0] ^ State[5] ^ State[10] ^ State[15] ^ State[20];
		BC[1] = State[1] ^ State[6] ^ State[11] ^ State[16] ^ State[21];
		BC[2] = State[2] ^ State[7] ^ State[12] ^ State[17] ^ State[22];
		BC[3] = State[3] ^ State[8] ^ State[13] ^ State[18] ^ State[23];
		BC[4] = State[4] ^ State[9] ^ State[14] ^ State[19] ^ State[24];

		Pho_Theta_function_No_use_Thread(0, 0, State[0], State[5], State[10], State[15], State[20]);
		Pho_Theta_function_No_use_Thread(1, 5, State[1], State[6], State[11], State[16], State[21]);
		Pho_Theta_function_No_use_Thread(2, 10, State[2], State[7], State[12], State[17], State[22]);
		Pho_Theta_function_No_use_Thread(3, 15, State[3], State[8], State[13], State[18], State[23]);
		Pho_Theta_function_No_use_Thread(4, 20, State[4], State[9], State[14], State[19], State[24]);

		Chi_lota_function_No_use_Thread(0, buffer[0], buffer[6], buffer[12], buffer[18], buffer[24]);
		Chi_lota_function_No_use_Thread(5, buffer[3], buffer[9], buffer[10], buffer[16], buffer[22]);
		Chi_lota_function_No_use_Thread(10, buffer[1], buffer[7], buffer[13], buffer[19], buffer[20]);
		Chi_lota_function_No_use_Thread(15, buffer[4], buffer[5], buffer[11], buffer[17], buffer[23]);
		Chi_lota_function_No_use_Thread(20, buffer[2], buffer[8], buffer[14], buffer[15], buffer[21]);
		State[0] ^= RC[j];
	}
}

void Keccak_Init(SHA3_INFO* info) {
	info->msglen = 0;
	memset(info->buffer, 0, 136);
	memset((byte*)State, 0, 200);
}

__device__ byte pi[25] =
{
	0, 6, 12, 18, 24,
	3, 9, 10, 16, 22,
	1, 7, 13, 19, 20,
	4, 5, 11, 17, 23,
	2, 8, 14, 15, 21
};

__device__ byte offset[Round + 1] = {
0, 36, 3, 41, 18,
1, 44, 10, 45, 2,
62, 6, 43, 15, 61,
28, 55, 25, 21, 56,
27, 20, 39, 8, 14
};

__global__ void GPU_Keccak(ulong* state, ulong* BC)
{
	__shared__ ulong temp[25];


	int tid = threadIdx.x;
	ulong temp1 = BC[(tid + 4) % 5] ^ (ROTL64((BC[tid + 1] % 5), 1));

	temp[tid]		= ROTL64(state[tid]			^ temp1, offset[tid * 5]);
	temp[tid + 5]	= ROTL64(state[tid + 5]		^ temp1, offset[tid * 5 + 1]);
	temp[tid + 10]	= ROTL64(state[tid + 10]	^ temp1, offset[tid * 5 + 2]);
	temp[tid + 15]	= ROTL64(state[tid + 15]	^ temp1, offset[tid * 5 + 3]);
	temp[tid + 20]	= ROTL64(state[tid + 20]	^ temp1, offset[tid * 5 + 4]);

	//__syncthreads();

	state[5 * tid] = ((temp[pi[tid]]) ^ ((~temp[pi[tid + 1]] & temp[pi[tid + 2]])));
	state[5 * tid + 1] = ((temp[pi[tid + 1]]) ^ ((~temp[pi[tid + 2]] & temp[pi[tid + 3]])));
	state[5 * tid + 2] = ((temp[pi[tid + 2]]) ^ ((~temp[pi[tid + 3]] & temp[pi[tid + 4]])));
	state[5 * tid + 3] = ((temp[pi[tid + 3]]) ^ ((~temp[pi[tid + 4]] & temp[pi[tid + 0]])));
	state[5 * tid + 4] = ((temp[pi[tid + 4]]) ^ ((~temp[pi[tid]] & temp[pi[tid + 1]])));

	//__syncthreads();
}

void GPU_Absorbing(SHA3_INFO* info, byte* pt, word msglen){
	info->msglen += msglen;
	ulong* gpu_state;
	ulong* gpu_BC;
	int* a = (int*)malloc(sizeof(int));
	int* dev_a;
	HANDLE_ERROR(hipMalloc((void**)&gpu_state, 25 * sizeof(ulong)));
	HANDLE_ERROR(hipMalloc((void**)&gpu_BC, 5 * sizeof(ulong)));
	HANDLE_ERROR(hipMalloc((void**)&dev_a, sizeof(int)));

	while (msglen >= 136)
	{
		memcpy(info->buffer, pt, msglen);
		for (int i = 0; i < 17; i++) {
			State[i] = State[i] ^ ENDIAN_CHANGE(*(ulong*)(info->buffer + (i << 3)));
		}

		for (int i = 0; i < Round; i++) {

			BC[0] = State[0] ^ State[5] ^ State[10] ^ State[15] ^ State[20];
			BC[1] = State[1] ^ State[6] ^ State[11] ^ State[16] ^ State[21];
			BC[2] = State[2] ^ State[7] ^ State[12] ^ State[17] ^ State[22];
			BC[3] = State[3] ^ State[8] ^ State[13] ^ State[18] ^ State[23];
			BC[4] = State[4] ^ State[9] ^ State[14] ^ State[19] ^ State[24];
			
			for (int j = 0; j < 25; j++)
				printf("%llx ", State[j]);
			printf("\n");
			HANDLE_ERROR(hipMemcpy(gpu_state, State, sizeof(ulong) * 25, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(gpu_BC, BC, sizeof(ulong) * 5, hipMemcpyHostToDevice));
			GPU_Keccak <<<1, 5 >>>(gpu_state, BC);

			printf("END\n");
			HANDLE_ERROR(hipMemcpy(State, gpu_state, sizeof(ulong) * 25, hipMemcpyDeviceToHost));
	
		}
		msglen -= 136;
		pt = pt + 136;
	}
	HANDLE_ERROR(hipFree(gpu_state));
	HANDLE_ERROR(hipFree(gpu_BC));
}

void GGPU_Keccak(byte* pt, word bytelen, byte* output)
{
	SHA3_INFO info;
	Keccak_Init(&info);
	GPU_Absorbing(&info, pt, bytelen);
	for (int i = 0; i < 4; i++)
		*(ulong*)(output + 8 * i) = ENDIAN_CHANGE(State[i]);
}

void KeccakR1088_Absorting_No_use(SHA3_INFO* info, byte* pt, word msglen)
{
	info->msglen += msglen;
	while (msglen >= 136) {
		memcpy(info->buffer, pt, 136);
		keecak_Function_NO_use_Thread(info);
		msglen -= 136;
		pt = pt + 136;
	}

	//PADDING
	memcpy(info->buffer, pt, msglen);
	word temp = info->msglen % 136;
	memset(info->buffer + temp, 0, 136 - temp);
	info->buffer[temp++] = 0x06;
	info->buffer[135] = 0x80;
	keecak_Function_NO_use_Thread(info);

}


void KeccakR1088_Sqeezing(SHA3_INFO* info, byte* output) {

	for (int i = 0; i < 4; i++)
		*(ulong*)(output + 8 * i) = ENDIAN_CHANGE(State[i]);
}



void KeccakR1088_No_use_Thread(byte* pt, word bytelen, byte* output)
{
	SHA3_INFO info;
	Keccak_Init(&info);
	KeccakR1088_Absorting_No_use(&info, pt, bytelen);
	KeccakR1088_Sqeezing(&info, output);
}

#endif